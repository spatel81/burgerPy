#include "hip/hip_runtime.h"
#include "kernel.h"
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <chrono>

#include <Python.h>
#define NPY_NO_DEPRECATED_API NPY_1_7_API_VERSION
#include <numpy/arrayobject.h>

#define TPB 64

//Laplacian
__global__ void ddKernel(double *d_out, const double *d_in, int size, double h) {
  const int i = threadIdx.x + blockDim.x*blockIdx.x;
  if (i >= size - 1 || i == 0) return;
  d_out[i] = (d_in[i - 1] - 2.f*d_in[i] + d_in[i + 1]) / (h*h);
}

//First Derivative
__global__ void dKernel(double *d_out, const double *d_in, int size, double h) {
  const int i = threadIdx.x + blockDim.x*blockIdx.x;
  if (i >= size - 1 || i == 0) return;
  d_out[i] = (d_in[i + 1] - d_in[i - 1]) / (2*h);
}

//Burger's Update
//Could get better performance using shared mem
__global__ void burgerUpdate_Kernel(double *d_out, const double *d_in, int size, double c1, double c2) {

  const int i = threadIdx.x + blockDim.x*blockIdx.x;
  if (i > 0 && i < size-1 ) { 
     d_out[i] = d_in[i] + (d_in[i - 1] - 2.f*d_in[i] + d_in[i + 1])*c1 - d_in[i]*(d_in[i + 1] - d_in[i - 1])*c2;
  } else if (i == 0) {
     d_out[i] = d_in[size-2];
  } else if (i >= size-1) {
     d_out[i]=d_in[1];
  }  

}

constexpr int N = 256; // number of points in spatial discretization
void PyIt(PyObject *p_func, double *u);
void pynalyze(PyObject *p_func);

int main() {
  
  /****Some python initialization****/
  Py_Initialize();
  PyRun_SimpleString("import sys");
  PyRun_SimpleString("sys.path.append(\".\")");
     std::cout << "Initialization of Python: Done" << std::endl;

  // initialize numpy array library
  import_array1(-1);

  PyObject* pName = PyUnicode_DecodeFSDefault("python_module"); // Python filename
  PyObject* pModule = PyImport_Import(pName);
  Py_DECREF(pName); // finished with this string so release reference
     std::cout << "Loaded python module" << std::endl;

  std::cout << "Loading functions from module" << std::endl;
  PyObject* pcollect = PyObject_GetAttrString(pModule, "collection_func");
  PyObject* py_PlotField = PyObject_GetAttrString(pModule, "analyses_plotField");
  PyObject* py_SVD = PyObject_GetAttrString(pModule, "analyses_SVD");

  Py_DECREF(pModule); // finished with this module so release reference
     std::cout << "Loaded functions" << std::endl;

  /***********************************/

  const double PI = 3.1415926;	
  const int Ntests = 100;
  const double h = 2.0*PI/N;
  const double dt = 0.001; 
  const double FT = 2.000;  //Final Time
  const double NU = 0.01;   //diffusion param

  double s1 = dt*NU / (h*h);
  double s2 = dt / (2.0*h);

  double uh[N+2] = { 0.0 };
  double uh_prev[N+2] = { 0.0 };
  double result_parallel[N+2] = { 0.0 };

  double x;
  //Initialize
  for (int i = 1; i < N+1; ++i) {
    x    = 2.0*(i-1)*PI/N;
    uh[i]      = sin(x);
    uh_prev[i] = sin(x);
  }
  uh[0]   = uh[N]; // Ghost Nodes
  uh[N+1] = uh[1]; // Ghost Nodes
  
  uh_prev[0]   = uh_prev[N]; // Ghost Nodes
  uh_prev[N+1] = uh_prev[1]; // Ghost Nodes

  //Set-up some pointers and allocate device memory
  double *ud=0, *ud_prev=0;
  hipMalloc(&ud, (N+2)*sizeof(double)); 
  hipMalloc(&ud_prev, (N+2)*sizeof(double));
  hipMemcpy(ud_prev, uh, (N+2)*sizeof(double), hipMemcpyHostToDevice);

  double t = 0.0;
  auto walltime_start = std::chrono::high_resolution_clock::now();
  do{

      //Do the Burger's update with FD  
      burgerUpdate_Kernel<<<(N + TPB - 1)/TPB, TPB>>>(ud, ud_prev, N+2, s1, s2);
      {
        PyIt(pcollect, ud); //collect to global python data array  
      }
  
      //Move the current solution to the previous timestep 
      hipMemcpy(ud_prev, ud, (N+2)*sizeof(double), hipMemcpyDeviceToDevice);
      std::cout << "time = " << t << std::endl;
      t = t + dt;  

  }while(t<FT);
  auto walltime_finish = std::chrono::high_resolution_clock::now();
  double wallTime = std::chrono::duration<double,std::milli>(walltime_finish-walltime_start).count(); 
  std::cout << "avg. solver wallTime : " << wallTime/Ntests << std::endl;
 
  //copy result to host 
  hipMemcpy(result_parallel, ud, N*sizeof(double), hipMemcpyDeviceToHost);
  hipFree(ud);
  hipFree(ud_prev);

      {
         Py_DECREF(pcollect);
      }

      //Plot the field
      {

        pynalyze(py_PlotField);  //collect to global python data array  
	Py_DECREF(py_PlotField);

      }

      //SVD
      {
	pynalyze(py_SVD); //Do Tensorflow stuff  
	Py_DECREF(py_SVD);
      }
}


//Python Wrappers
void PyIt(PyObject *p_func, double *u)
{
  PyObject* pArgs = PyTuple_New(1);

  //Numpy array dimensions
  npy_intp dim[] = {N+2};

  // create a new Python array that is a wrapper around u (not a copy) and put it in tuple pArgs
  PyObject* array_1d = PyArray_SimpleNewFromData(1, dim, NPY_FLOAT64, u);
  PyTuple_SetItem(pArgs, 0, array_1d);

  // pass array into our Python function and cast result to PyArrayObject
  PyArrayObject* pValue = (PyArrayObject*) PyObject_CallObject(p_func, pArgs);
  //std::cout << "Called python data collection function successfully"<<std::endl;

  Py_DECREF(pArgs);
  Py_DECREF(pValue);
  // We don't need to decref array_1d because PyTuple_SetItem steals a reference
}


void pynalyze(PyObject *p_func)
{
  // panalsyses_func doesn't require an argument so pass nullptr
  PyArrayObject* pValue = (PyArrayObject*)PyObject_CallObject(p_func, nullptr);
  std::cout << "Called python analyses function successfully"<<std::endl;

  Py_DECREF(pValue);
  // We don't need to decref array_1d because PyTuple_SetItem steals a reference
}
